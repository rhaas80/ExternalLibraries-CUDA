#include "hip/hip_runtime.h"
#include "cctk.h"
#include "cctk_Parameters.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void vector_add(float *out, float *a, float *b, int n)
{
  for(int i = 0; i < n; i++){
    out[i] = a[i] + b[i];
  }
}

extern "C"
int CUDA_Test(void) {
  DECLARE_CCTK_PARAMETERS;

  const int N = 10;
  float *a, *b, *out; 
  
  // Allocate memory
  a   = (float*)malloc(sizeof(float) * N);
  b   = (float*)malloc(sizeof(float) * N);
  out = (float*)malloc(sizeof(float) * N);
  for(int i = 0; i < N; i++) {
    a[i] = 1.;
    b[i] = 0.;
    out[i] = 42;
  }
  
  float *d_a, *d_b, *d_out;
  hipMalloc((void**)&d_a, sizeof(float) * N);
  hipMalloc((void**)&d_b, sizeof(float) * N);
  hipMalloc((void**)&d_out, sizeof(float) * N);
  
  hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
  
  vector_add<<<1,1>>>(d_out, d_a, d_b, N);
  hipError_t err = hipGetLastError();
  
  hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyDeviceToHost);
  if(err == hipSuccess) {
    if(out[0] != a[0] + b[0]) {
      CCTK_VERROR("CUDA kernel produced incorrrect result %f != %f", out[0],
                  a[0] + b[0]);
    }
  } else {
    CCTK_VERROR("CUDA failed with %s", hipGetErrorString(err));
  }
   
  hipFree((void**)&d_a);
  hipFree((void**)&d_b);
  hipFree((void**)&d_out);
  free(out);
  free(b);
  free(a);

  return 0;
} 
